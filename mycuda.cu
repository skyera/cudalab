#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include "doctest.h"
#define ANKERL_NANOBENCH_IMPLEMENT
#include "nanobench.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define N 10000000
#define MAX_ERR 1e-6

void vector_add(float *out, float *a, float *b, int n) {
    for (int i = 0; i < n; i++) {
        out[i] = a[i] + b[i];
    }
}

__global__ void cuda_vector_add(float *out, float *a, float *b, int n) {
    for (int i = 0; i < n; i++) {
        out[i] = a[i] + b[i];
    }
}

__global__ void cuda_vector_add2(float *out, float *a, float *b, int n) {
    int index = 0;
    int stride = 1;

    for (int i = index; i < n; i += stride) {
        out[i] = a[i] + b[i];
    }
}

__global__ void cuda_vector_add3(float *out, float *a, float *b, int n) {
    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int i = index; i < n; i += stride) {
        out[i] = a[i] + b[i];
    }
}

__global__ void cuda_vector_add_grid(float *out, float *a, float *b, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        out[tid] = a[tid] + b[tid];
    }
}

void add(int n, float *x, float *y) {
    for (int i = 0; i < n; i++) {
        y[i] = x[i] + y[i];
    }
}

__global__
void cuda_add(int n, float *x, float *y) {
    for (int i = 0; i < n; ++i) {
        y[i] = x[i] + y[i];
    }
}

__global__
void cuda_add_thread(int n, float *x, float *y) {
    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int i = index; i < n; i+= stride) {
        y[i] = x[i] + y[i];
    }
}

TEST_CASE("vector_add") {
    float *a, *b, *out;

    a = (float*) malloc(sizeof(float) * N);
    b = (float*) malloc(sizeof(float) * N);
    out = (float*) malloc(sizeof(float) * N);

    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }
    vector_add(out, a, b, N);
}

TEST_CASE("cuda_vector_add") {
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;
    hipError_t e;

    a = (float*) malloc(sizeof(float) * N);
    b = (float*) malloc(sizeof(float) * N);
    out = (float*) malloc(sizeof(float) * N);

    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    e = hipMalloc((void**)&d_a, sizeof(float) * N);
    REQUIRE(e == hipSuccess);
    e = hipMalloc((void**)&d_b, sizeof(float) * N);
    REQUIRE(e == hipSuccess);
    e = hipMalloc((void**)&d_out, sizeof(float) * N);
    REQUIRE(e == hipSuccess);

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    cuda_vector_add<<<1,1>>>(d_out, d_a, d_b, N);

    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    
    for (int i = 0; i < N; i++) {
        INFO("i = ", i, " out=", out[i], " a=", a[i], " b=", b[i]);
        REQUIRE(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }
    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    free(a);
    free(b);
    free(out);
}

TEST_CASE("cuda_vector_addi_mthreads") {
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;
    hipError_t e;

    a = (float*) malloc(sizeof(float) * N);
    b = (float*) malloc(sizeof(float) * N);
    out = (float*) malloc(sizeof(float) * N);

    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    e = hipMalloc((void**)&d_a, sizeof(float) * N);
    REQUIRE(e == hipSuccess);
    e = hipMalloc((void**)&d_b, sizeof(float) * N);
    REQUIRE(e == hipSuccess);
    e = hipMalloc((void**)&d_out, sizeof(float) * N);
    REQUIRE(e == hipSuccess);

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    cuda_vector_add3<<<1,256>>>(d_out, d_a, d_b, N);

    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    
    for (int i = 0; i < N; i++) {
        INFO("i = ", i, " out=", out[i], " a=", a[i], " b=", b[i]);
        REQUIRE(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }
    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    free(a);
    free(b);
    free(out);
}

void run_cuda_add_grid()
{
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;
    hipError_t e;

    a = (float*) malloc(sizeof(float) * N);
    b = (float*) malloc(sizeof(float) * N);
    out = (float*) malloc(sizeof(float) * N);

    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    e = hipMalloc((void**)&d_a, sizeof(float) * N);
    REQUIRE(e == hipSuccess);
    e = hipMalloc((void**)&d_b, sizeof(float) * N);
    REQUIRE(e == hipSuccess);
    e = hipMalloc((void**)&d_out, sizeof(float) * N);
    REQUIRE(e == hipSuccess);

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    
    /* int block_size = 256; */
    /* int grid_size = (N + block_size) / block_size; */
    int grid_size = 2;
    int block_size = 32;
    cuda_vector_add_grid<<<grid_size,block_size>>>(d_out, d_a, d_b, N);

    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    
    for (int i = 0; i < N; i++) {
        INFO("i = ", i, " out=", out[i], " a=", a[i], " b=", b[i]);
        REQUIRE(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }
    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    free(a);
    free(b);
    free(out);
}

TEST_CASE("cuda_vector_add_grid") {
    ankerl::nanobench::Bench bench;

    bench.run("add_grid", [&] {
        run_cuda_add_grid();
    });    
}

void run_add() {
#undef N
    int N = 1 << 20;

    float *x = new float[N];
    float *y = new float[N];

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    add(N, x, y);

    float max_error = 0.0f;
    for (int i = 0; i < N; i++) {
        max_error = fmax(max_error, fabs(y[i]-3.0f));
    }
    std::cout << "max error: " << max_error << std::endl;  
    
    delete [] x;
    delete [] y;
}

TEST_CASE("add") {
    ankerl::nanobench::Bench bench;

    bench.run("add", [&] {
        run_add();
    });    
}

void run_cuda_add()
{
#undef N
    int N = 1 << 20;

    float *x, *y;
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    cuda_add<<<1,1>>>(N, x, y);
    hipDeviceSynchronize();

    float max_error = 0.0f;
    for (int i = 0; i < N; i++) {
        max_error = fmax(max_error, fabs(y[i]-3.0f));
    }
    std::cout << "max error: " << max_error << std::endl;  
    
    hipFree(x);
    hipFree(y);
}

TEST_CASE("cuda_add") {
    ankerl::nanobench::Bench bench;
    
    bench.run("yyy", [&] {
        run_add();
    });
}

void run_add_thread()
{
#undef N
    int N = 1 << 20;

    float *x, *y;
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    cuda_add_thread<<<1,256>>>(N, x, y);
    hipDeviceSynchronize();

    float max_error = 0.0f;
    for (int i = 0; i < N; i++) {
        max_error = fmax(max_error, fabs(y[i]-3.0f));
    }
    std::cout << "max error: " << max_error << std::endl;  
    
    hipFree(x);
    hipFree(y);
}

TEST_CASE("cuda_add_thread") {
    ankerl::nanobench::Bench bench;
    
    bench.run("xxx", [&] {
        run_add_thread();
    });
}

TEST_CASE("bench") {
    double d = 1.0;
    ankerl::nanobench::Bench().run("some double ops", [&] {
        d += 1.0/d;
        if (d > 5.0) {
            d-= 5.0;
        }
        ankerl::nanobench::doNotOptimizeAway(d);
    });
}

__global__ void cuda_hello()
{
    printf("Hello world from GPU\n");
}

TEST_CASE("hello") {
    cuda_hello<<<1,1>>>();
}

TEST_CASE("device_count") {
    int count = 0;
    hipError_t e = hipGetDeviceCount(&count);
    REQUIRE(e == hipSuccess);
    REQUIRE(count == 1);
    std::cout << "device count: " << count << "\n";
}

TEST_CASE("compute_mode") {
    int compute_mode = -1;
    int curr_dev = 0;
    
    hipError_t e = hipDeviceGetAttribute(&compute_mode,
            hipDeviceAttributeComputeMode, curr_dev);
    REQUIRE(e == hipSuccess);
    std::cout << "compute mode: " << compute_mode << "\n";
}

TEST_CASE("major_minor") {
    int major = 0;
    int minor = 0;
    int curr_dev = 0;
    
    hipError_t e = hipDeviceGetAttribute(&major,
            hipDeviceAttributeComputeCapabilityMajor, curr_dev);
    REQUIRE(e == hipSuccess);
    std::cout << "major: " << major << "\n";

    e = hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor,
            curr_dev);
    REQUIRE(e == hipSuccess);
    std::cout << "minor: " << minor << "\n";
}

TEST_CASE("getdevice") {
    int dev=0;
    
    hipError_t e = hipGetDevice(&dev);
    REQUIRE(e == hipSuccess);
    std::cout << "dev " << dev << "\n";
}

void print_devprop(const hipDeviceProp_t& devprop) {
    std::cout << "name: " << devprop.name << "\n"
        << "ECCEnabled: " << devprop.ECCEnabled << "\n"
        << "clockRate: " << devprop.clockRate << "\n"
        << "l2CacheSize: " << devprop.l2CacheSize << "\n";
}

TEST_CASE("devprop") {
    int dev = 0;
    hipDeviceProp_t devprop;

    hipError_t e = hipGetDeviceProperties(&devprop, dev);
    REQUIRE(e == hipSuccess);
    print_devprop(devprop);
}

__global__ void increment_kernel(int *g_data, int inc_value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    g_data[idx] = g_data[idx] + inc_value;
}

TEST_CASE("asyncAPI") {
    int n = 16 * 1024 * 1024;
    int nbytes = n * sizeof(int);
    int *a = 0;

    hipError_t e = hipHostMalloc((void**)&a, nbytes);
    REQUIRE(e == hipSuccess);
    memset(a, 0, nbytes);

    int *d_a = 0;
    e = hipMalloc((void**)&d_a, nbytes);
    REQUIRE(e == hipSuccess);
    e = hipMemset(d_a, 255, nbytes);
    REQUIRE(e == hipSuccess);

    // event
    hipEvent_t start, stop;
    e = hipEventCreate(&start);
    REQUIRE(e == hipSuccess);
    e = hipEventCreate(&stop);
    REQUIRE(e == hipSuccess);
    
    hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, 0);

    dim3 threads = dim3(512, 1);
    dim3 blocks = dim3(n / threads.x, 1);
    int value = 26;
    
    increment_kernel<<<blocks, threads, 0, 0>>>(d_a, value);
    hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, 0);

    e = hipHostFree(a);
    REQUIRE(e == hipSuccess);
    e = hipFree(d_a);
    REQUIRE(e == hipSuccess);
}
